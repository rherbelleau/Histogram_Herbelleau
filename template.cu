#include "hip/hip_runtime.h"
// Includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <unistd.h>

// Includes, CUDA
#include <hip/hip_runtime.h>

// Includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h> // helper functions for SDK examples
#include "common.h"

// Define constants
#define line_max 200000
#define char_max 40
#define ascii_nb 128
#define threads_per_block 1024
#define borne_inf 32
#define borne_sup 126
#define first_up 65
#define last_up 90

////////////////////////////////////////////////////////////////////////////////
// Declarations
////////////////////////////////////////////////////////////////////////////////
void Histogram(char* inputFileName, char* outputFileName);

void writeOutputCSV(unsigned long long result[ascii_nb], char* outputFileName);

void processBatchInKernel(  char** d_data,
                            char h_data[line_max][char_max],
                            int nbLine,
                            size_t pitch,
                            int lineSize,
                            unsigned long long** d_result,
                            int resultSize,
                            unsigned long long resultStorage[ascii_nb]);

////////////////////////////////////////////////////////////////////////////////
//! Kernel function to execute the computation in threads using only Global Memory
//! @param d_data  input data in global memory
//! @param d_result  output result as array in global memory
//! @param nbLine  input size of the data in global memory
//! @param pitch  input pitch size of in the data global memory
////////////////////////////////////////////////////////////////////////////////

__global__ 
void kernelHistoGlobal(char* d_data, unsigned long long* d_result, int nbLine, size_t pitch) {
    
    const unsigned int tidb = threadIdx.x;
    const unsigned int ti = blockIdx.x*blockDim.x + tidb;
    unsigned long long unit = 1;
    
    // Each thread compute a single line of the data
    if (ti < nbLine) {
		char* line = (char *)((char*)d_data + ti * pitch);
		int index = 0;
		int currentLetter = line[index];

        // Each char is converted to int and adds a unit to the corresponding index in the global memory
		while (currentLetter > 0) {
	    	atomicAdd(&d_result[currentLetter], unit);
	    	index++;
	    	currentLetter = line[index];
		}
    }
}
                            

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
	
    // Process the arguments of the call
	int c;
	char *inputFileName = NULL;
	char *outputFileName = NULL;
	while ((c = getopt (argc, argv, "i:o:h")) != -1)
		switch(c) {
			case 'i':
				inputFileName = optarg;
				break;
			case 'o':
				outputFileName = optarg;
				break;
		}

	printf("\n%s Starting...\n\n", argv[0]);

    // Start timer
    StopWatchInterface *timer = 0;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

    // Do the computation
	Histogram(inputFileName, outputFileName);

    // Stop timer
    sdkStopTimer(&timer);
    printf("Processing time: %f (ms)\n\n", sdkGetTimerValue(&timer));
    sdkDeleteTimer(&timer);

    printf("Input file processed successfully.\n");
    if (outputFileName) {
        printf("Check results in %s.\n\n", outputFileName);
    } else {
        printf("Check results in out.csv.\n\n");
    }

	exit(EXIT_SUCCESS);
}

////////////////////////////////////////////////////////////////////////////////
//! Generate the Histogram
//! @param inputFileName input name of the file to process
//! @param outputFileName input name of the file to output the histogram in
////////////////////////////////////////////////////////////////////////////////
void Histogram(char* inputFileName, char* outputFileName) {
    
    // Compute result and data sizes
    unsigned int resultSize = ascii_nb * sizeof(unsigned long long);
    unsigned int lineSize = char_max * sizeof(char);

    // Load input file
    FILE *inputFile = NULL;
    inputFile = fopen(inputFileName, "r");
    if (!inputFile) {
        printf("Wrong input file\n");
		exit(EXIT_FAILURE);
    }

    // Allocate device memory
    char* d_data;
    unsigned long long* d_result;
    size_t pitch;
    checkCudaErrors(hipMallocPitch((void **) &d_data, &pitch, lineSize, line_max));
    checkCudaErrors(hipMalloc((void **) &d_result, resultSize));

    // Allocate host memory
    char h_data[line_max][char_max];
    unsigned long long resultStorage[ascii_nb];
    char str[char_max];
    int nbLine = 0;
    int batchNum = 1;
    
    // Iterate over the file's lines
    while (fgets(str, char_max, inputFile)) {
	
        // Batch size reached, send data to kernel for process
		if (nbLine == line_max) {

            printf("Batch N°%i: %i lines. \n", batchNum, nbLine);
	    	processBatchInKernel(&d_data, h_data, nbLine, pitch, lineSize, &d_result, resultSize, resultStorage);
            
            nbLine = 0;
            batchNum++;
		}

        // Add current line to the Batch
        strcpy(h_data[nbLine], str);
        nbLine++;
    }
    
    // Process last Batch (< line_max lines)
    printf("Batch N°%i: %i lines. \n", batchNum, nbLine);
    processBatchInKernel(&d_data, h_data, nbLine, pitch, lineSize, &d_result, resultSize, resultStorage);
    
    fclose(inputFile);
    
    // Write the output
    writeOutputCSV(resultStorage, outputFileName);

    // Cleanup memory
    checkCudaErrors(hipFree(d_data));
    checkCudaErrors(hipFree(d_result));
}

////////////////////////////////////////////////////////////////////////////////
//! Send batch data to kernel and store the output in resultStorage
//! @param d_data input pointer to the allocated memory for the input data on the device
//! @param h_data input the list of strings to process
//! @param nbLine input number of lines to process for the current batch
//! @param pitch input pitch size of the array in the device 
//! @param lineSize input size of a single line
//! @param d_result input pointer to the allo
//! @param resultSize input pointer to the allocated memory for the output data on the device
//! @param resultStorage output result of the computation as an array
////////////////////////////////////////////////////////////////////////////////

void processBatchInKernel(  char** d_data,
                            char h_data[line_max][char_max],
                            int nbLine,
                            size_t pitch,
                            int lineSize,
                            unsigned long long** d_result,
                            int resultSize,
                            unsigned long long resultStorage[ascii_nb]) {
    // Allocate host memory for result
    unsigned long long h_result[ascii_nb];

    // Setup execution parameters
    dim3  grid((nbLine + threads_per_block - 1) / threads_per_block, 1, 1);
    dim3  threads(threads_per_block, 1, 1);

    // Copy data to device
    checkCudaErrors(hipMemcpy2D(*d_data, pitch, h_data, lineSize, lineSize, line_max, hipMemcpyHostToDevice));
    
    // Execute the kernel
    kernelHistoGlobal<<< grid, threads, 0 >>>(*d_data, *d_result, nbLine, pitch);

    getLastCudaError("Kernel execution failed");
    
    // Copy result from device to host
    checkCudaErrors(hipMemcpy(&h_result, *d_result, resultSize, hipMemcpyDeviceToHost));

    // Copy the result into resultStorage
    for (int index = 0; index < ascii_nb; index++) {
        resultStorage[index] = h_result[index];
    }
}

////////////////////////////////////////////////////////////////////////////////
//! Write the given output to the CSV file
//! @param result input the given ouput of the computations as an array of int
//! @param outputFileName input file name to write in
////////////////////////////////////////////////////////////////////////////////

void writeOutputCSV(unsigned long long result[ascii_nb], char* outputFileName) {

    // Load output file
	FILE *outputFile;
	char asciiChar;
    if (outputFileName) {
        outputFile = fopen(outputFileName, "w+");
    } else {
        outputFile = fopen("outputHisto.csv", "w+");
    }
	
    // Write the result
	for (int index = borne_inf; index <= borne_sup; index++) {

        if (index >= first_up && index <= last_up) {
            // Add uppercase count to char count
            result[index + borne_inf] += result[index];
        } else {
            // Write count in file
            asciiChar = index;
		    fprintf(outputFile, "%c: %llu\n", asciiChar, result[index]);
        }

	}

	fclose(outputFile);
}


